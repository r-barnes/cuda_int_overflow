
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <type_traits>


// CUDA API error handler
#define CUDA_CHECK(call) \
    if ((call) != hipSuccess) { \
        const auto err = hipGetLastError(); \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(1); \
    }

template <typename T>
class MySafeInt {
    static_assert(std::is_same_v<T, uint32_t> || std::is_same_v<T, int32_t>,
                  "MySafeInt can only hold uint32_t or int32_t");
    T value = 0;

public:
    constexpr MySafeInt() = default;
    __device__ constexpr explicit MySafeInt(T v) : value(v) {}

    __device__ constexpr operator T() const { return value; }

    // Arithmetic operations (except multiplication between MySafeInt types)
    __device__  constexpr MySafeInt operator+(const MySafeInt& rhs) const { return MySafeInt(value + rhs.value); }
    __device__  constexpr MySafeInt operator*(const MySafeInt& rhs) const { return MySafeInt(value * rhs.value); }
    // __device__  constexpr MySafeInt operator-(const MySafeInt& rhs) const { return MySafeInt(value - rhs.value); }
    // __device__  constexpr MySafeInt operator/(const MySafeInt& rhs) const { return MySafeInt(value / rhs.value); }
    // __device__  constexpr MySafeInt operator%(const MySafeInt& rhs) const { return MySafeInt(value % rhs.value); }

    // Multiplication is only allowed with fundamental types, never with MySafeInt
    // template <typename U, typename = std::enable_if_t<std::is_arithmetic_v<U>>>
    // __device__ constexpr MySafeInt operator*(U rhs) const { return MySafeInt(value * rhs); }

    // Compound assignment operators
    // __device__ constexpr MySafeInt& operator+=(const MySafeInt& rhs) { value += rhs.value; return *this; }
    // __device__ constexpr MySafeInt& operator-=(const MySafeInt& rhs) { value -= rhs.value; return *this; }
    // __device__ constexpr MySafeInt& operator/=(const MySafeInt& rhs) { value /= rhs.value; return *this; }
    // __device__ constexpr MySafeInt& operator%=(const MySafeInt& rhs) { value %= rhs.value; return *this; }

    // template <typename U, typename = std::enable_if_t<std::is_arithmetic_v<U>>>
    // __device__ constexpr MySafeInt& operator*=(U rhs) { value *= rhs; return *this; }

    // Increment and decrement operators
    // __device__ constexpr MySafeInt& operator++() { ++value; return *this; }
    // __device__ constexpr MySafeInt operator++(int) { MySafeInt tmp(*this); ++value; return tmp; }
    // __device__ constexpr MySafeInt& operator--() { --value; return *this; }
    // __device__ constexpr MySafeInt operator--(int) { MySafeInt tmp(*this); --value; return tmp; }

    // Comparison operators
    // __device__ constexpr bool operator==(const MySafeInt& rhs) const { return value == rhs.value; }
    // __device__ constexpr bool operator!=(const MySafeInt& rhs) const { return value != rhs.value; }
    __device__ constexpr bool operator<(const MySafeInt& rhs) const { return value < rhs.value; }
    __device__ constexpr bool operator<(const int32_t o) const { return value < o; }
    __device__ constexpr bool operator<(const uint32_t o) const { return value < o; }
    // __device__ constexpr bool operator<=(const MySafeInt& rhs) const { return value <= rhs.value; }
    // __device__ constexpr bool operator>(const MySafeInt& rhs) const { return value > rhs.value; }
    // __device__ constexpr bool operator>=(const MySafeInt& rhs) const { return value >= rhs.value; }
};

// Non-member multiplication operators for fundamental types
// template <typename T, typename U, typename = std::enable_if_t<std::is_arithmetic_v<U>>>
// __device__ constexpr MySafeInt<T> operator*(U lhs, const MySafeInt<T>& rhs) {
//     return MySafeInt<T>(lhs * static_cast<T>(rhs));
// }

// Explicit non-member operator[] to enable array indexing
// template <typename T, typename U>
// constexpr auto& operator[](T* ptr, const MySafeInt<U>& idx) {
//     return ptr[idx.get()];
// }

// template <typename T, typename U>
// constexpr const auto& operator[](const T* ptr, const MySafeInt<U>& idx) {
//     return ptr[idx.get()];
// }


// Kernel function to multiply two arrays
__global__ void multiply(uint32_t *a, uint32_t *b, uint32_t *c, uint32_t N) {
  const auto idx = MySafeInt<uint32_t>(blockIdx.x) * MySafeInt<uint32_t>(blockDim.x) + MySafeInt<uint32_t>(threadIdx.x);
  if (idx < N) {
      c[idx] = a[idx] * b[idx];
  }
}

int main() {
  // Size of the arrays
  uint32_t N = 1000;

  std::mt19937 rng;
  // rng.seed(std::random_device()());
  std::uniform_int_distribution<std::mt19937::result_type> dist(0, std::numeric_limits<uint32_t>::max());

  // Host arrays
  std::vector<uint32_t> a(N);
  std::vector<uint32_t> b(N);
  std::vector<uint32_t> c(N);

  // Fill a and b with random numbers between 0 and INT_MAX
  for (int i = 0; i < N; i++) {
      a[i] = dist(rng);
      b[i] = dist(rng);
  }

  // Device arrays
  uint32_t *d_a, *d_b, *d_c;
  CUDA_CHECK(hipMalloc(&d_a, N * sizeof(uint32_t)));
  CUDA_CHECK(hipMalloc(&d_b, N * sizeof(uint32_t)));
  CUDA_CHECK(hipMalloc(&d_c, N * sizeof(uint32_t)));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_a, a.data(), N * sizeof(uint32_t), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, b.data(), N * sizeof(uint32_t), hipMemcpyHostToDevice));

  // Launch the kernel
  const int threadsPerBlock = 256;
  const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  multiply<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
  // Check that the kernel launched successfully
  CUDA_CHECK(hipPeekAtLastError());

  // Copy data from device to host
  CUDA_CHECK(hipMemcpy(c.data(), d_c, N * sizeof(uint32_t), hipMemcpyDeviceToHost));

  // Print the result
  for (int i = 0; i < N; i++) {
      std::cout << c[i] << " ";
  }
  std::cout << std::endl;

  // Free device memory
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));

  return 0;
}